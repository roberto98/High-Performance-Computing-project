/*
Per compilare:
nvcc progetto_cuda.cu
time ./a.out img.data
*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <complex>
#include <cmath>

// Ranges of the set
#define MIN_X -2
#define MAX_X 1
#define MIN_Y -1
#define MAX_Y 1

// Image ratio
#define RATIO_X (MAX_X - MIN_X)
#define RATIO_Y (MAX_Y - MIN_Y)

// Image size
#define RESOLUTION 1000
#define WIDTH (RATIO_X * RESOLUTION)
#define HEIGHT (RATIO_Y * RESOLUTION)

#define STEP ((double)RATIO_X / WIDTH)

#define DEGREE 2        // Degree of the polynomial
#define ITERATIONS 1000 // Maximum number of iterations

using namespace std;

/*
// PSEUDOCODICE PRESO DA WIKIPEDIA
__global__ void mandelbrotKernel(int *image, double step, double minX, double minY, int width, int height, int iterations)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos < width * height)
    {       
        image[pos] = 0;

        const int row = pos / width;
        const int col = pos % width;

        double x0 = col * step + minX; 
        double y0 = row * step + minY;

        double x = 0.0;
        double y = 0.0;
        int iteration = 1;
        while (x * x + y * y <= 4 && iteration < iterations)
        {
            double xtemp = x * x - y * y + x0;
            y = 2 * x * y + y0;
            x = xtemp;
            iteration++;
        }

        if(iteration != ITERATIONS)
            image[pos] = iteration;
    }
}
*/

__global__ void mandelbrotKernel(int *image, double step, double minX, double minY, int width, int height, int iterations)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos < width * height)
    {       
        image[pos] = 0;
        const int row = pos / width;
        const int col = pos % width;
        double c_real = col * STEP + MIN_X;
        double c_imm = row * STEP + MIN_Y;
        double z_real = 0;
        double z_imm = 0;
        double z_square_real;
        double z_square_imm;

        for(int i=1; i<=ITERATIONS; i++)
        {
            z_square_real = z_real*z_real - z_imm*z_imm;
            z_square_imm = 2 * z_real * z_imm;
            z_real = z_square_real + c_real;
            z_imm = z_square_imm + c_imm;
            if( z_real*z_real + z_imm*z_imm >= 4){
                image[pos] = i;
                break;
            }
        }
    }
}


int main(int argc, char **argv)
{
    int *const image = new int[HEIGHT * WIDTH];
    printf("L'immagine ha dimensione %d\n", HEIGHT * WIDTH);

    // ----------------------- ALLOCATION --------------------- //
    int *d_image;
    hipMalloc(&d_image, sizeof(int) * WIDTH * HEIGHT); // Allocate memory for the result on the device
    //cudaMemcpy(d_image, image, sizeof(int) * WIDTH * HEIGHT, cudaMemcpyHostToDevice); // Copy data from host to device

    // ---------------------- CREATE TIMER ------------------ //
    hipEvent_t start_time, stop_time;
    hipEventCreate(&start_time);
    hipEventCreate(&stop_time);

    // --------------------- THREADS & BLOCKS ----------------- //
    dim3 threadsPerBlock(1024);
    dim3 numBlocks((WIDTH * HEIGHT + threadsPerBlock.x-1) / threadsPerBlock.x); 

    printf("threadsPerBlock.x: %u \n", threadsPerBlock.x);
    printf("numBlocks.x: %u \n", numBlocks.x);

    hipEventRecord(start_time);
    mandelbrotKernel<<<numBlocks, threadsPerBlock>>>(d_image, STEP, MIN_X, MIN_Y, WIDTH, HEIGHT, ITERATIONS);
    hipDeviceSynchronize(); // Wait for the kernel to finish
    hipMemcpy(image, d_image, sizeof(int) * WIDTH * HEIGHT, hipMemcpyDeviceToHost); // Copy data from device to the host
    
    // ---------------------- STOP TIMER ------------------ //
    hipEventRecord(stop_time);
    hipEventSynchronize(stop_time);

    float elapsed_ref = 0;
    hipEventElapsedTime(&elapsed_ref, start_time, stop_time);
    printf("Time elapsed: %f milliseconds\n", elapsed_ref) ;

    // ---------------------- FRATTALE ------------------ //
    ofstream matrix_out; // Write the result to a file
 
    if (argc < 2)
    {
        cout << "Please specify the output file as a parameter." << endl;
        return -1;
    }

    matrix_out.open(argv[1], ios::trunc);
    if (!matrix_out.is_open())
    {
        cout << "Unable to open file." << endl;
        return -2;
    }

    for (int row = 0; row < HEIGHT; row++)
    {
        for (int col = 0; col < WIDTH; col++)
        {
            matrix_out << image[row * WIDTH + col];

            if (col < WIDTH - 1)
                matrix_out << ',';
        }
        if (row < HEIGHT - 1)
            matrix_out << endl;
    }
    matrix_out.close();

    delete[] image;
    hipFree(d_image);
    return 0;
}